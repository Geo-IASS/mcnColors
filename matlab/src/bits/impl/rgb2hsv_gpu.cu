#include "hip/hip_runtime.h"
// @file rgb2hsv_gpu.cu
// @brief RGB2HSV 
// @author Samuel Albanie

/*
Copyright (C) 2017- Samuel Albanie.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "rgb2hsv_impl.hpp"
#include <bits/datacu.hpp>
#include <bits/mexutils.h>
#include <bits/data.hpp>
#include <assert.h>
#include <float.h>
#include <cstdio>
#include <math.h>
#include <string.h>

/* ------------------------------------------------------------ */
/*                                                      kernels */
/* ------------------------------------------------------------ */

template<typename T> __global__ void
rgb2hsv_kernel(T* output,
              const T* data,
              const int volume,
              bool* valid_range, 
              const int height,
              const int width,
              const int size)
{
  int hsvIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (hsvIndex < volume) {
    int depth = 3 ; // RGB input
    int area = height * width ;
    int s = hsvIndex % area ;  // spatial offset
    int b = hsvIndex / (area * depth) ; // batch element
    int rIdx = (b * area * 3) + s ;
    int gIdx = rIdx + area ;
    int bIdx = rIdx + 2 * area ;
    int c = (hsvIndex / area) % depth ;

    T R = data[rIdx] ;
    T G = data[gIdx] ;
    T B = data[bIdx] ;

    // check input ranges
    bool valid_R_range = R <= 1 && R >= 0 ;
    bool valid_G_range = G <= 1 && G >= 0 ;
    bool valid_B_range = B <= 1 && B >= 0 ;
    if (!(valid_R_range && valid_G_range && valid_B_range)) {
      valid_range[0] = 1 ;
     }
   
    T out ;
    T maxRGB = max(R, max(G, B)) ;
    T minRGB = min(R, min(G, B)) ;
    T delta = maxRGB - minRGB ;
    switch (c) { // H, S or V output
      case 0: // Compute hue
        out = data[rIdx] ;
        if (R == maxRGB) {
          out = (G - B) / delta ;
        } else if (G == maxRGB) {
          out = 2 + ( B - R ) / delta ;
        } else { // B max
          out = 4 + ( R - G ) / delta ;
        } 
        out = out / 6 ; // use [0,1], rather than 360 degrees
        if (out < 0) {
          out = out + 1 ; 
        }
        break ;
      case 1: // compute saturation
        if (maxRGB == 0) {
          out = 0 ; // follow matlab convention
        } else {
          out = delta / maxRGB ;
        }
        break ;
      case 2: // compute value
        out = maxRGB ; // store value
        break ;
    }
    output[hsvIndex] = out ;
  }
}



namespace vl { namespace impl {

/* ------------------------------------------------------------ */
/*                                                      rgb2hsv */
/* ------------------------------------------------------------ */

    template<typename T>
    struct rgb2hsv<vl::VLDT_GPU,T>
    {

    static vl::ErrorCode
    forward(Context& context, 
            T* output, 
            T const* data,
            size_t height, 
            size_t width, 
            size_t size) 
{    
    int volume = height * width * 3 * size ;

    // set flag for input checking
    bool* valid_range ;
    hipMalloc( (void **) &valid_range, sizeof(bool)) ;
    hipMemset(valid_range, 0, sizeof(bool)) ; // init to zero

    rgb2hsv_kernel<T><<< vl::divideAndRoundUp(volume, 
      VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>(output, data, 
        volume, valid_range, height, width, size) ;

    bool* h_valid_range = new bool[1] ;
    hipMemcpy(h_valid_range, valid_range, sizeof(bool), hipMemcpyDeviceToHost) ;
    hipError_t status = hipPeekAtLastError() ;
    // TODO: clean up error handling here
    // currently the input validation is done on the device to prevent a speed 
    // overhead, but requires a slightly ungainlly use of error codes.
    if ((status != hipSuccess) || (h_valid_range[0] != 0)) {
        if (h_valid_range[0] != 0) {
          mexPrintf("invalid RGB input values (must lie in [0,1]) \n") ;
        }
        return vl::VLE_Cuda ;
    } else {
      return vl::VLE_Success ;
    }
   }
 } ;

/* ------------------------------------------------------------ */
/*                                                      hsv2rgb */
/* ------------------------------------------------------------ */

    template<typename T>
    struct hsv2rgb<vl::VLDT_GPU,T>
    {

    static vl::ErrorCode
    forward(Context& context, 
            T* output, 
            T const* data,
            size_t height, 
            size_t width, 
            size_t size) 
{    
    int volume = height * width * 3 * size ;

    // set flag for input checking
    bool* valid_range ;
    hipMalloc( (void **) &valid_range, sizeof(bool)) ;
    hipMemset(valid_range, 0, sizeof(bool)) ; // init to zero

    rgb2hsv_kernel<T><<< vl::divideAndRoundUp(volume, 
      VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>(output, data, 
        volume, valid_range, height, width, size) ;

    bool* h_valid_range = new bool[1] ;
    hipMemcpy(h_valid_range, valid_range, sizeof(bool), hipMemcpyDeviceToHost) ;
    hipError_t status = hipPeekAtLastError() ;
    // TODO: clean up error handling here
    // currently the input validation is done on the device to prevent a speed 
    // overhead, but requires a slightly ungainlly use of error codes.
    if ((status != hipSuccess) || (h_valid_range[0] != 0)) {
        if (h_valid_range[0] != 0) {
          mexPrintf("invalid RGB input values (must lie in [0,1]) \n") ;
        }
        return vl::VLE_Cuda ;
    } else {
      return vl::VLE_Success ;
    }
   }
 } ;
} } // namespace vl::impl

template struct vl::impl::rgb2hsv<vl::VLDT_GPU, float> ;
template struct vl::impl::hsv2rgb<vl::VLDT_GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::rgb2hsv<vl::VLDT_GPU, double> ;
template struct vl::impl::hsv2rgb<vl::VLDT_GPU, double> ;
#endif
