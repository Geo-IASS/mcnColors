#include "hip/hip_runtime.h"
// @file rgb2hsv_gpu.cu
// @brief RGB2HSV 
// @author Samuel Albanie

/*
Copyright (C) 2017- Samuel Albanie.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "rgb2hsv_impl.hpp"
#include <bits/datacu.hpp>
#include <bits/mexutils.h>
#include <bits/data.hpp>
#include <assert.h>
#include <float.h>
#include <cstdio>
#include <math.h>
#include <string.h>

/* ------------------------------------------------------------ */
/*                                                      kernels */
/* ------------------------------------------------------------ */

template<typename T> __global__ void
rgb2hsv_kernel(T* output,
              const T* data,
              const int volume,
              bool* valid_range, 
              const int height,
              const int width,
              const int size)
{
  int hsvIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (hsvIndex < volume) {
    int depth = 3 ; // RGB input
    int area = height * width ;
    int s = hsvIndex % area ;  // spatial offset
    int b = hsvIndex / (area * depth) ; // batch element
    int rIdx = (b * area * 3) + s ;
    int gIdx = rIdx + area ;
    int bIdx = rIdx + 2 * area ;
    int c = (hsvIndex / area) % depth ;

    T R = data[rIdx] ;
    T G = data[gIdx] ;
    T B = data[bIdx] ;

    // check input ranges
    bool valid_R_range = R <= 1 && R >= 0 ;
    bool valid_G_range = G <= 1 && G >= 0 ;
    bool valid_B_range = B <= 1 && B >= 0 ;
    if (!(valid_R_range && valid_G_range && valid_B_range)) {
      valid_range[0] = 1 ;
     }
   
    T out ;
    T maxRGB = max(R, max(G, B)) ;
    T minRGB = min(R, min(G, B)) ;
    T delta = maxRGB - minRGB ;
    switch (c) { // H, S or V output
      case 0: // Compute hue
        out = data[rIdx] ;
        if (R == maxRGB) {
          out = (G - B) / delta ;
        } else if (G == maxRGB) {
          out = 2 + ( B - R ) / delta ;
        } else { // B max
          out = 4 + ( R - G ) / delta ;
        } 
        out = out / 6 ; // use [0,1], rather than 360 degrees
        if (out < 0) {
          out = out + 1 ; 
        }
        break ;
      case 1: // compute saturation
        if (maxRGB == 0) {
          out = 0 ; // follow matlab convention
        } else {
          out = delta / maxRGB ;
        }
        break ;
      case 2: // compute value
        out = maxRGB ; // store value
        break ;
    }
    output[hsvIndex] = out ;
  }
}

template<typename T> __global__ void
hsv2rgb_kernel(T* output,
              const T* data,
              const int volume,
              bool* valid_range, 
              const int height,
              const int width,
              const int size)
{
  int rgbIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (rgbIndex < volume) {
    int depth = 3 ; // HSV input
    int area = height * width ;
    int s = rgbIndex % area ;  // spatial offset
    int b = rgbIndex / (area * depth) ; // batch element
    int hIdx = (b * area * 3) + s ;
    int sIdx = hIdx + area ;
    int vIdx = hIdx + 2 * area ;
    int c = (rgbIndex / area) % depth ;

    T H = data[hIdx] ;
    T S = data[sIdx] ;
    T V = data[vIdx] ;

    // check input ranges
    bool valid_H_range = H <= 1 && H >= 0 ;
    bool valid_S_range = S <= 1 && S >= 0 ;
    bool valid_V_range = V <= 1 && V >= 0 ;
    if (!(valid_H_range && valid_S_range && valid_V_range)) {
      valid_range[0] = 1 ;
     }
    T out ;
    T H_ = H * 6 ; // follow standard convention for Hue computation
    int cRegion = (int) trunc(H_) ; // map into one of six color regions
    T rem = H_ - cRegion ; // store remainder

    // incorporate the post addition of (V - chroma) into quantities
    // that can be assigned directly
    T Q1 = V * (1 - S) ;
    T Q2 = V * (1 - (S * rem)) ;
    T Q3 = V * (1 - (S * (1 - rem))) ;

    switch (cRegion) {
      case 0:
        switch (c) { // RGB switch
          case 0: out = V ; break ;  
          case 1: out = Q3 ; break ;  
          case 2: out = Q1 ; break ;  
        } break ;
      case 1:
        switch (c) { // RGB switch
          case 0: out = Q2 ; break ; 
          case 1: out = V ; break ; 
          case 2: out = Q1 ; break ;  
        } break ;
      case 2:
        switch (c) { // RGB switch
          case 0: out = Q1 ; break ; 
          case 1: out = V ; break ; 
          case 2: out = Q3 ; break ;  
        } break ;
      case 3:
        switch (c) { // RGB switch
          case 0: out = Q1 ; break ; 
          case 1: out = Q2 ; break ; 
          case 2: out = V ; break ;  
        } break ;
      case 4:
        switch (c) { // RGB switch
          case 0: out = Q3 ; break ; 
          case 1: out = Q1 ; break ; 
          case 2: out = V ; break ;  
        } break ;
      case 5:
        switch (c) { // RGB switch
          case 0: out = V ; break ; 
          case 1: out = Q1 ; break ; 
          case 2: out = Q2 ; break ;  
        } break ;
      case 6: // match MATLAB convention here
        switch (c) { // RGB switch
          case 0: out = V ; break ;  
          case 1: out = Q3 ; break ;  
          case 2: out = Q1 ; break ;  
        } break ;
    }
    output[rgbIndex] = out ;
  }
}



namespace vl { namespace impl {

/* ------------------------------------------------------------ */
/*                                                      rgb2hsv */
/* ------------------------------------------------------------ */

    template<typename T>
    struct rgb2hsv<vl::VLDT_GPU,T>
    {

    static vl::ErrorCode
    forward(Context& context, 
            T* output, 
            T const* data,
            size_t height, 
            size_t width, 
            size_t size) 
{    
    int volume = height * width * 3 * size ;

    // set flag for input checking
    bool* valid_range ;
    hipMalloc( (void **) &valid_range, sizeof(bool)) ;
    hipMemset(valid_range, 0, sizeof(bool)) ; // init to zero

    rgb2hsv_kernel<T><<< vl::divideAndRoundUp(volume, 
      VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>(output, data, 
        volume, valid_range, height, width, size) ;

    bool* h_valid_range = new bool[1] ;
    hipMemcpy(h_valid_range, valid_range, sizeof(bool), hipMemcpyDeviceToHost) ;
    hipError_t status = hipPeekAtLastError() ;
    // TODO: clean up error handling here
    // currently the input validation is done on the device to prevent a speed 
    // overhead, but requires a slightly ungainlly use of error codes.
    if ((status != hipSuccess) || (h_valid_range[0] != 0)) {
        if (h_valid_range[0] != 0) {
          mexPrintf("invalid RGB input values (must lie in [0,1]) \n") ;
        }
        return vl::VLE_Cuda ;
    } else {
      return vl::VLE_Success ;
    }
   }
 } ;

/* ------------------------------------------------------------ */
/*                                                      hsv2rgb */
/* ------------------------------------------------------------ */

    template<typename T>
    struct hsv2rgb<vl::VLDT_GPU,T>
    {

    static vl::ErrorCode
    forward(Context& context, 
            T* output, 
            T const* data,
            size_t height, 
            size_t width, 
            size_t size) 
{    
    int volume = height * width * 3 * size ;

    // set flag for input checking
    bool* valid_range ;
    hipMalloc( (void **) &valid_range, sizeof(bool)) ;
    hipMemset(valid_range, 0, sizeof(bool)) ; // init to zero

    hsv2rgb_kernel<T><<< vl::divideAndRoundUp(volume, 
      VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>(output, data, 
        volume, valid_range, height, width, size) ;

    bool* h_valid_range = new bool[1] ;
    hipMemcpy(h_valid_range, valid_range, sizeof(bool), hipMemcpyDeviceToHost) ;
    hipError_t status = hipPeekAtLastError() ;
    // TODO: clean up error handling here
    // currently the input validation is done on the device to prevent a speed 
    // overhead, but requires a slightly ungainlly use of error codes.
    if ((status != hipSuccess) || (h_valid_range[0] != 0)) {
        if (h_valid_range[0] != 0) {
          mexPrintf("invalid RGB input values (must lie in [0,1]) \n") ;
        }
        return vl::VLE_Cuda ;
    } else {
      return vl::VLE_Success ;
    }
   }
 } ;
} } // namespace vl::impl

template struct vl::impl::rgb2hsv<vl::VLDT_GPU, float> ;
template struct vl::impl::hsv2rgb<vl::VLDT_GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::rgb2hsv<vl::VLDT_GPU, double> ;
template struct vl::impl::hsv2rgb<vl::VLDT_GPU, double> ;
#endif
